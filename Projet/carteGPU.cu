#include "hip/hip_runtime.h"
#include "utils/commonCUDA.hpp"
#include "utils/chronoGPU.hpp"

#include "carteCPU.hpp"


__global__ void kernelMap(int *data,const Heightmap h_in, Heightmap h_out, const int Cx,const int Cy)
{
    for(int indexY = blockDim.y * blockIdx.y + threadIdx.y ; indexY < h_in.getHeight(); indexY += blockDim.y * gridDim.y)
    {
        for(int indexX = blockDim.x * blockIdx.x + threadIdx.x; indexX < h_in.getWidth(); indexX += blockDim.x * gridDim.x)
        {
        // DDA entre le point c (Cx, Cy) et le point P (indexX, indexY);

            float Dx, Dy, D;
            float Cx_dda = (float) Cx, Cy_dda = (float) Cy;
            Dx = indexX - Cx;
            Dy = indexY - Cy;

            D = max(abs(Dx), abs(Dy));

            float incX = Dx / D;
            float incY = Dy / D;
            for (int i = 0; i < D - 1; i++)
            {
                Cx_dda += incX;
                Cy_dda += incY;
                data[i * 2] = (int)round(Cx_dda);
                data[i * 2 + 1] = (int)round(Cy_dda);
            }

        // Fin calcule DDA

        // Calcule Angle  
            float Dz;
            Dz = h_in.getPixel(indexX, indexY) - h_in.getPixel(Cx, Cy);
            double angle_ref = atan(Dz / sqrt((Dx * Dx) + (Dy * Dy)));
            double angle;

            h_out.setPixel(indexX, indexY, 244);
            for (int i = 0; i < (D - 1); i++)
            {
                Dx = indexX - data[i * 2];
                Dy = indexY - data[i * 2 + 1];
                Dz = h_in.getPixel(indexX, indexY) - h_in.getPixel(data[i * 2], data[i * 2 + 1]);  
                angle = atan(Dz / sqrt((Dx * Dx) + (Dy * Dy)));

                if (angle_ref >= angle)
                {
                    h_out.setPixel(indexX, indexY, 0);
                    break;
                }  
            }        
        }
    }
    h_out.saveTo("img/Result/CPU/LimousinCPU.ppm");
}



void carteGPU(int *data, const Heightmap h_in, Heightmap h_out,const int Cx,const int Cy)
{
    int *dev_data;
    Heightmap *dev_h_in;
    Heightmap *dev_h_out;

    //hipMalloc(&dev_data, )


}