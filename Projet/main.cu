#include <iostream>
#include <cstdlib>
#include <iomanip>

#include "utils/chronoCPU.hpp"
#include "utils/chronoGPU.hpp"
#include "los/ppm.hpp"
#include "carteCPU.hpp"

int main(int argc, char **argv)
{
	// Parse program arguments
	// ================================================================================================================
	// Allocation and initialization

	// ================================================================================================================

	// ================================================================================================================
	// CPU sequential
	std::cout << "============================================" << std::endl;
	std::cout << "         Sequential version on CPU          " << std::endl;
	std::cout << "============================================" << std::endl;


	std::vector<int> dda(10000, 0);

    Heightmap h_in("img/input/1.input.ppm");
	uint8_t * buffer = h_in.getPtr();
	size_t buffer_size = h_in.getHeight() * h_in.getWidth();

	/*for(int i = 0; i < 100; i++)
	{
		std::cout << +buffer[i] << " : " << +h_data.getPixel(i, 0) << std::endl;
	}*/

    Heightmap h_out(h_in.getWidth(), h_in.getHeight());

	ChronoCPU chrCPU;
	chrCPU.start();		// CPU method
	drawMap(dda.data(), h_in.getPtr(), h_out.getPtr(), h_in.getWidth() * h_in.getHeight(), h_in.getWidth(), h_in.getHeight(),  245, 497);
	//void drawMap(int *data,uint8_t *h_data, Heightmap h_out, const int MapSize, const int MapWidth, const int MapHeight,const int Cx,const int Cy)
	h_out.saveTo("img/Result/CPU/LimousinCPU.ppm");
	chrCPU.stop();

	const float timeComputeCPU = chrCPU.elapsedTime();
	std::cout << "-> Done : " << std::fixed << std::setprecision(2) << timeComputeCPU << " ms" << std::endl
			  << std::endl;

	// ================================================================================================================

	// ================================================================================================================
	// GPU CUDA
	std::cout << "============================================" << std::endl;
	std::cout << "          Parallel version on GPU           " << std::endl;
	std::cout << "============================================" << std::endl;

	// GPU allocation
	ChronoGPU chrGPU;
	chrCPU.start();	

	chrCPU.stop();
	// ======================

	const float timeAllocGPU = chrCPU.elapsedTime();
	std::cout << "-> Done : " << std::fixed << std::setprecision(2) << timeAllocGPU << " ms" << std::endl
			  << std::endl;

	// Copy from host to device
	std::cout << "Copying data from Host to Device" << std::endl;
	chrGPU.start();

	chrGPU.stop();

	const float timeHtoDGPU = chrGPU.elapsedTime();
	std::cout << "-> Done : " << timeHtoDGPU << " ms" << std::endl
			  << std::endl;


	// Launch kernel
	std::cout << "Summming vectors" << std::endl;
	chrGPU.start();	// GPU method

	chrGPU.stop();

	const float timeComputeGPU = chrGPU.elapsedTime();
	std::cout << "-> Done : " << std::fixed << std::setprecision(2) << timeComputeGPU << " ms" << std::endl
			  << std::endl;

	// copy from device to host
	std::cout << "Copying data from Device to Host" << std::endl;

	chrGPU.start();

	chrGPU.stop();
	const float timeDtoHGPU = chrGPU.elapsedTime();
	std::cout << "-> Done : " << std::fixed << std::setprecision(2) << timeDtoHGPU << " ms" << std::endl
			  << std::endl;

	// Free GPU memory

	// ================================================================================================================

	std::cout << "============================================" << std::endl;
	std::cout << "              Checking results              " << std::endl;
	std::cout << "============================================" << std::endl;



	std::cout << "Congratulations! Job's done!" << std::endl
			  << std::endl;

	std::cout << "============================================" << std::endl;
	std::cout << "            Times recapitulation            " << std::endl;
	std::cout << "============================================" << std::endl;
	std::cout << "-> CPU	Sequential" << std::endl;
	std::cout << "   - Computation:    " << std::fixed << std::setprecision(2)
			  << timeComputeCPU << " ms" << std::endl;
	std::cout << "-> GPU	" << std::endl;
	std::cout << "   - Allocation:     " << std::fixed << std::setprecision(2)
			  << timeAllocGPU << " ms " << std::endl;
	std::cout << "   - Host to Device: " << std::fixed << std::setprecision(2)
			  << timeHtoDGPU << " ms" << std::endl;
	std::cout << "   - Computation:    " << std::fixed << std::setprecision(2)
			  << timeComputeGPU << " ms" << std::endl;
	std::cout << "   - Device to Host: " << std::fixed << std::setprecision(2)
			  << timeDtoHGPU << " ms " << std::endl
			  << std::endl;

	return EXIT_SUCCESS;
}